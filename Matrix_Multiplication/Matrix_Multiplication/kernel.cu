#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void matMul_kernel_f32(
    float* output,
    const float* input_a,
    const float* input_b,
    int M, int K, int N,
    const int tcount)
{
    int pos = threadIdx.x + blockIdx.x * blockDim.x;
    if (pos >= tcount) return;

    int w_idx = pos % N;
    int h_idx = pos / N;

    output[h_idx * N + w_idx] = 0.f;

    for (int k = 0; k < K; ++k) {
        output[h_idx * N + w_idx] += input_a[h_idx * K + k] * input_b[k * N + w_idx];
    }
}


void matmult_cu(int M, int N, int K, float* mat_a, float* mat_b, float* mat_c)
{
    std::cout << "==================================================" << std::endl;
    std::cout << "[INFO] CUDA Matrix Multiplication" << std::endl;

    //device-side data
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_o = 0;

    // allocate device memory
    CUDA_CHECK(hipMalloc((void**)&dev_a, M * K * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&dev_b, K * N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&dev_o, M * N * sizeof(int)));

    uint64_t dur_time = 0;
    uint64_t start_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

    for (int i = 0; i < ITERS; i++) {

        //copy from host to device 
        CUDA_CHECK(hipMemcpy(dev_a, mat_a, M * K * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dev_b, mat_b, K * N * sizeof(float), hipMemcpyHostToDevice));

        //launch a kernel on the GPU with one thread for each element.
        int thread_cnt = M * N;
        int block = 256;
        int grid = ((thread_cnt - 1) / block + 1);

        dim3 dimGrid(grid, 1, 1);
        dim3 dimBlock(block, 1, 1);

        uint64_t start_time2 = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

        matMul_kernel_f32 << <dimGrid, dimBlock >> > (dev_o, dev_a, dev_b, M, K, N, thread_cnt);

        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipPeekAtLastError());

        uint64_t end_time2 = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        dur_time += (end_time2 - start_time2);

        //copy from device to host
        CUDA_CHECK(hipMemcpy(mat_c, dev_o, M * N * sizeof(int), hipMemcpyDeviceToHost));
    }
    uint64_t end_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

    // ��� ���
    std::cout << "[INFO] Avg elapsed time = " << (end_time - start_time) / (ITERS) << " [milliseconds] (with data transfer time)" << std::endl;
    std::cout << "[INFO] Avg elapsed time = " << (dur_time) / (ITERS) << " [milliseconds] (without data transfer time)" << std::endl;
    std::cout << "==================================================" << std::endl;
    //free device memory
    CUDA_CHECK(hipFree(dev_a));
    CUDA_CHECK(hipFree(dev_b));
    CUDA_CHECK(hipFree(dev_o));
}

#define TILE_WIDTH 32

__global__ void matMul_kernel_shared_memory_rect(
    float* output,
    const float* input_a,
    const float* input_b,
    int M, int K, int N,
    const int tcount)
{
    extern __shared__ float s_A[];
    extern __shared__ float s_B[];
    int pos = threadIdx.x + blockIdx.x * blockDim.x;
    if (pos >= tcount) return;
    int col = pos % N;
    int row = pos / N;
    int tx = pos % TILE_WIDTH;
    int ty = col / TILE_WIDTH;
    float sum = 0.f;

    for (int t = 0; t < (K - 1) / TILE_WIDTH + 1; t++)
    {
        if (row < M && t * TILE_WIDTH + tx < N)
            s_A[ty * TILE_WIDTH + tx] = input_a[row * K + t * TILE_WIDTH + tx];
        else
            s_A[ty * TILE_WIDTH + tx] = 0.0;
        if (t * TILE_WIDTH + ty < N && col < N)
            s_B[ty * TILE_WIDTH + tx] = input_b[(t * TILE_WIDTH + ty) * K + col];
        else
            s_B[ty * TILE_WIDTH + tx] = 0.0;

        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; i++)
            sum += s_A[ty * TILE_WIDTH + i] * s_B[i * TILE_WIDTH + tx];
        __syncthreads();
    }

    if (row < M && col < N)
        output[col + row * N] = sum;
}

void matmult_cu_shared(int M, int N, int K, float* mat_a, float* mat_b, float* mat_c)
{
    std::cout << "==================================================" << std::endl;
    std::cout << "[INFO] CUDA Matrix Multiplication with shared memory" << std::endl;

    //device-side data
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_o = 0;

    // allocate device memory
    CUDA_CHECK(hipMalloc((void**)&dev_a, M * K * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&dev_b, K * N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&dev_o, M * N * sizeof(int)));

    uint64_t dur_time = 0;
    uint64_t start_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

    for (int i = 0; i < ITERS; i++) {

        //copy from host to device 
        CUDA_CHECK(hipMemcpy(dev_a, mat_a, M * K * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dev_b, mat_b, K * N * sizeof(float), hipMemcpyHostToDevice));

        //launch a kernel on the GPU with one thread for each element.
        int thread_cnt = M * N;
        const int GRID_WIDTH = (M * N - 1) / (TILE_WIDTH * TILE_WIDTH) + 1;
        dim3 dimGrid(GRID_WIDTH, 1, 1);
        dim3 dimBlock(TILE_WIDTH * TILE_WIDTH, 1, 1);

        uint64_t start_time2 = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

        matMul_kernel_f32 << <dimGrid, dimBlock, TILE_WIDTH * TILE_WIDTH * sizeof(float) >> > (dev_o, dev_a, dev_b, M, K, N, thread_cnt);

        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipPeekAtLastError());

        uint64_t end_time2 = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        dur_time += (end_time2 - start_time2);

        //copy from device to host
        CUDA_CHECK(hipMemcpy(mat_c, dev_o, M * N * sizeof(int), hipMemcpyDeviceToHost));
    }
    uint64_t end_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

    // ��� ���
    std::cout << "[INFO] Avg elapsed time = " << (end_time - start_time) / (ITERS) << " [milliseconds] (with data transfer time)" << std::endl;
    std::cout << "[INFO] Avg elapsed time = " << (dur_time) / (ITERS) << " [milliseconds] (without data transfer time)" << std::endl;
    std::cout << "==================================================" << std::endl;
    //free device memory
    CUDA_CHECK(hipFree(dev_a));
    CUDA_CHECK(hipFree(dev_b));
    CUDA_CHECK(hipFree(dev_o));
}

hipblasStatus_t Sgemm(hipblasHandle_t Blas, hipblasOperation_t AOp, hipblasOperation_t BOp,
    const float* dev_A, int WidthA, int HeightA,
    const float* dev_B, int WidthB, int HeightB,
    float *dev_C, float Alpha = 1.0f, float Beta = 0.0f)
{
    int lda = WidthA;
    int ldb = WidthB;

    if (AOp != HIPBLAS_OP_N) {
        int tmp = WidthA;
        WidthA = HeightA;
        HeightA = tmp;
    }
    if (BOp != HIPBLAS_OP_N) {
        int tmp = WidthB;
        WidthB = HeightB;
        HeightB = tmp;
    }
    int m = WidthB;
    int n = HeightA;
    int k = WidthA;

    return hipblasSgemm(Blas, BOp, AOp, m, n, k, &Alpha, dev_B, ldb, dev_A, lda, &Beta, dev_C, m);
}

void matmult_cublas(int M, int N, int K, float* mat_a, float* mat_b, float* mat_c)
{
    std::cout << "==================================================" << std::endl;
    std::cout << "[INFO] Cublas Matrix Multiplication" << std::endl;

    hipblasHandle_t cublasHandle;
    CUDA_CHECK(hipblasCreate(&cublasHandle));

    //device-side data
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_o = 0;

    // allocate device memory
    CUDA_CHECK(hipMalloc((void**)&dev_a, M * K * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&dev_b, K * N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&dev_o, M * N * sizeof(int)));

    uint64_t dur_time = 0;
    uint64_t start_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

    for (int i = 0; i < ITERS; i++) {

        //copy from host to device 
        CUDA_CHECK(hipMemcpy(dev_a, mat_a, M * K * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dev_b, mat_b, K * N * sizeof(float), hipMemcpyHostToDevice));

        //launch a kernel on the GPU with one thread for each element.
        uint64_t start_time2 = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

        Sgemm(cublasHandle, (hipblasOperation_t)0, (hipblasOperation_t)0, dev_a, K, M, dev_b, N, K, dev_o);

        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipPeekAtLastError());

        uint64_t end_time2 = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        dur_time += (end_time2 - start_time2);

        //copy from device to host
        CUDA_CHECK(hipMemcpy(mat_c, dev_o, M * N * sizeof(int), hipMemcpyDeviceToHost));
    }
    uint64_t end_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

    // ��� ���
    std::cout << "[INFO] Avg elapsed time = " << (end_time - start_time) / (ITERS) << " [milliseconds] (with data transfer time)" << std::endl;
    std::cout << "[INFO] Avg elapsed time = " << (dur_time) / (ITERS) << " [milliseconds] (without data transfer time)" << std::endl;
    std::cout << "==================================================" << std::endl;
    //free device memory
    CUDA_CHECK(hipFree(dev_a));
    CUDA_CHECK(hipFree(dev_b));
    CUDA_CHECK(hipFree(dev_o));
}